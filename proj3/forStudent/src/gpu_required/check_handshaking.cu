/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	/** YOUR CODE GOES BELOW **/
	int num_threads = blockDim.x * gridDim.x;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	for (int i = tid; i < numNodes; i+= num_threads) 
	{
		if(matches[i] == -1){
			if(strongNeighbor[strongNeighbor[i]] == i){ 
				if( matches[strongNeighbor[i]] == -1){
					matches[i] = strongNeighbor[i];
					matches[strongNeighbor[i]] = i;
				}		
			}
		}
	}
	/** YOUR CODE GOES ABOVE **/
}
