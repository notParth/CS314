/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void markFilterEdges_gpu(int * src, int * dst, int * matches, int * keepEdges, int numEdges) {
	/** YOUR CODE GOES BELOW **/
	int num_threads = blockDim.x * gridDim.x;;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	for (int i = tid; i < numEdges; i+=num_threads) 
	{
		if(matches[src[i]] == -1 && matches[dst[i]] == -1)
			keepEdges[i] = 1;
		else
			keepEdges[i] = 0;
	}
	/** YOUR CODE GOES ABOVE **/
}
