/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void exclusive_prefix_sum_gpu(int * oldSum, int * newSum, int distance, int numElements) {
	/** YOUR CODE GOES BELOW **/
	int num_threads = blockDim.x * gridDim.x;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	for(int i = tid; i < numElements; i+=num_threads)
	{
		if (distance == 0)
		{
			if(i==0)
				newSum[0] = 0;
			else
				newSum[i] = oldSum[i - 1];
		}
	
		else	
		{
	
			if(i >= distance)
				newSum[i] = oldSum[i - distance] + oldSum[i];
			else
				newSum[i] = oldSum[i];
		}
	}
	
	/** YOUR CODE GOES ABOVE **/
}
